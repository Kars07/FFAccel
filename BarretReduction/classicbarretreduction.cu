#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <nvtx3/nvToolsExt.h>

// Data types for finite field elements
typedef uint32_t fp_t;     // Assumes p fits in 32 bits
typedef uint64_t fp_wide_t; // Wide type for intermediate calculations

// Device constants
__device__ __constant__ fp_t PRIME_P;
__device__ __constant__ fp_t MU;

// Error checking macro
#define CUDA_CHECK(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(1); \
    } \
} while(0)

// Barrett reduction implementation 
__device__ __forceinline__ fp_t barrett_reduce(fp_wide_t x) {
    
    const int m = 32; // bit length of prime p (assuming 32-bit prime)
    fp_wide_t c = x >> (m - 1);
    
    fp_wide_t temp = c * MU;
    fp_wide_t quot = temp >> (m + 1);
    
    fp_wide_t rem = x - quot * PRIME_P;
    
    if (rem >= PRIME_P) {
        
        rem = rem - PRIME_P;

        if (rem >= PRIME_P) {

            rem = rem - PRIME_P;
        }
    }
    
    return (fp_t)rem;
}

// Basic finite field operations using Barrett reduction
__device__ __forceinline__ fp_t fp_add_barrett(fp_t a, fp_t b) {
    fp_wide_t sum = (fp_wide_t)a + b;
    return (sum >= PRIME_P) ? (sum - PRIME_P) : (fp_t)sum;
}

__device__ __forceinline__ fp_t fp_sub_barrett(fp_t a, fp_t b) {
    return (a >= b) ? (a - b) : (a + PRIME_P - b);
}

__device__ __forceinline__ fp_t fp_mul_barrett(fp_t a, fp_t b) {
    fp_wide_t prod = (fp_wide_t)a * b;
    return barrett_reduce(prod);
}

__device__ __forceinline__ fp_t fp_neg_barrett(fp_t a) {
    return (a == 0) ? 0 : (PRIME_P - a);
}

// Modular exponentiation using Barrett reduction
__device__ fp_t fp_pow_barrett(fp_t base, fp_t exp) {
    fp_t result = 1;
    base = barrett_reduce(base); // Ensure base is within the field
    
    while (exp > 0) {
        if (exp & 1) {
            result = fp_mul_barrett(result, base);
        }
        exp >>= 1;
        base = fp_mul_barrett(base, base);
    }
    return result;
}

// Extended Euclidean Algorithm for modular inverse
__device__ fp_t fp_inv_barrett(fp_t a) {
    if (a == 0) {
        return 0; // Invalid input
    }

    int64_t old_r = PRIME_P, r = a;
    int64_t old_s = 0, s = 1;

    while (r != 0) {
        int64_t quotient = old_r / r;
        
        int64_t temp = r;
        r = old_r - quotient * r;
        old_r = temp;

        temp = s;
        s = old_s - quotient * s;
        old_s = temp;
    }
    
    return (old_s < 0) ? (fp_t)(old_s + PRIME_P) : (fp_t)old_s;
}

// Fermat's Little Theorem for modular inverse using Barrett reduction
__device__ fp_t fp_inv_fermat_barrett(fp_t a) {
    if (a == 0) {
        return 0; // Invalid input
    }
    return fp_pow_barrett(a, PRIME_P - 2);
}

__device__ __forceinline__ fp_t fp_div_barrett(fp_t a, fp_t b) {
    return fp_mul_barrett(a, fp_inv_barrett(b));
}

// Batch operations kernels using Barrett reduction
__global__ void fp_add_arrays_barrett(const fp_t* a, const fp_t* b, fp_t* result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = fp_add_barrett(a[idx], b[idx]);
    }
}

__global__ void fp_sub_arrays_barrett(const fp_t* a, const fp_t* b, fp_t* result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = fp_sub_barrett(a[idx], b[idx]);
    }
}

__global__ void fp_mul_arrays_barrett(const fp_t* a, const fp_t* b, fp_t* result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = fp_mul_barrett(a[idx], b[idx]);
    }
}

__global__ void fp_pow_arrays_barrett(const fp_t* base, const fp_t* exp, fp_t* result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = fp_pow_barrett(base[idx], exp[idx]);
    }
}

__global__ void fp_inv_arrays_barrett(const fp_t* a, fp_t* result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = fp_inv_barrett(a[idx]);
    }
}

// Polynomial evaluation using Barrett reduction
__global__ void fp_poly_eval_barrett(
    const fp_t* coeffs, int degree,
    const fp_t* x_vals, fp_t* results, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        fp_t x = x_vals[idx];
        fp_t result = coeffs[degree];

        for (int i = degree - 1; i >= 0; i--) {
            result = fp_add_barrett(fp_mul_barrett(result, x), coeffs[i]);
        }
        results[idx] = result; 
    }
}

// Matrix multiplication using Barrett reduction
__global__ void fp_matrix_mul_barrett(const fp_t* A, const fp_t* B, fp_t* C,
                                     int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < k) {
        fp_t sum = 0;
        for (int i = 0; i < n; i++) {
            sum = fp_add_barrett(sum, fp_mul_barrett(A[row * n + i], B[i * k + col]));
        }
        C[row * k + col] = sum;
    }
}

// Host wrapper class with Barrett reduction
class FiniteFieldFpBarrett {
private:
    fp_t p;
    fp_t mu; // Precomputed Barrett constant
    
    // Compute Barrett constant μ = ⌊2^(2m)/p⌋
    fp_t compute_mu(fp_t prime) {
        // For 32-bit prime, 2m = 64
        // μ = ⌊2^64/p⌋
        // Using integer division to compute this
        uint64_t two_pow_64 = 0xFFFFFFFFFFFFFFFFULL; // 2^64 - 1 (approximation)
        return (fp_t)(two_pow_64 / prime);
    }
    
public:
    FiniteFieldFpBarrett(fp_t prime) : p(prime) {
        mu = compute_mu(prime);
        
        // Copy prime and mu to device constant memory
        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(PRIME_P), &p, sizeof(fp_t)));
        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(MU), &mu, sizeof(fp_t)));
        
        printf("Barrett reduction initialized with p = %u, μ = %u\n", p, mu);
    }
    
    void add_arrays(const fp_t* h_a, const fp_t* h_b, fp_t* h_result, int n) {
        nvtxRangePush("add_arrays_barrett");
        fp_t *d_a, *d_b, *d_result;

        nvtxRangePush("Memory Allocation");
        CUDA_CHECK(hipMalloc(&d_a, n * sizeof(fp_t)));
        CUDA_CHECK(hipMalloc(&d_b, n * sizeof(fp_t)));
        CUDA_CHECK(hipMalloc(&d_result, n * sizeof(fp_t)));
        nvtxRangePop();
        
        nvtxRangePush("Memory Copy H2D");
        CUDA_CHECK(hipMemcpy(d_a, h_a, n * sizeof(fp_t), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_b, h_b, n * sizeof(fp_t), hipMemcpyHostToDevice));
        nvtxRangePop();
        
        nvtxRangePush("Kernel Execution");
        int block_size = 256;
        int grid_size = (n + block_size - 1) / block_size;
        fp_add_arrays_barrett<<<grid_size, block_size>>>(d_a, d_b, d_result, n);
        CUDA_CHECK(hipDeviceSynchronize());
        nvtxRangePop();
       
        nvtxRangePush("Memory Copy D2H");
        CUDA_CHECK(hipMemcpy(h_result, d_result, n * sizeof(fp_t), hipMemcpyDeviceToHost));
        nvtxRangePop();
        
        nvtxRangePush("Memory Deallocation");
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_result);
        nvtxRangePop();
        nvtxRangePop();
    }
    
    void mul_arrays(const fp_t* h_a, const fp_t* h_b, fp_t* h_result, int n) {
        nvtxRangePush("mul_arrays_barrett");
        fp_t *d_a, *d_b, *d_result;
        
        nvtxRangePush("Memory Allocation");
        CUDA_CHECK(hipMalloc(&d_a, n * sizeof(fp_t)));
        CUDA_CHECK(hipMalloc(&d_b, n * sizeof(fp_t)));
        CUDA_CHECK(hipMalloc(&d_result, n * sizeof(fp_t)));
        nvtxRangePop();
        
        nvtxRangePush("Memory Copy H2D");
        CUDA_CHECK(hipMemcpy(d_a, h_a, n * sizeof(fp_t), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_b, h_b, n * sizeof(fp_t), hipMemcpyHostToDevice));
        nvtxRangePop();
        
        nvtxRangePush("Kernel Execution");
        int block_size = 256;
        int grid_size = (n + block_size - 1) / block_size;
        fp_mul_arrays_barrett<<<grid_size, block_size>>>(d_a, d_b, d_result, n);
        CUDA_CHECK(hipDeviceSynchronize());
        nvtxRangePop();
        
        nvtxRangePush("Memory Copy D2H");
        CUDA_CHECK(hipMemcpy(h_result, d_result, n * sizeof(fp_t), hipMemcpyDeviceToHost));
        nvtxRangePop();
        
        nvtxRangePush("Memory Deallocation");
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_result);
        nvtxRangePop();
        nvtxRangePop();
    }
    
    void inv_arrays(const fp_t* h_a, fp_t* h_result, int n) {
        nvtxRangePush("inv_arrays_barrett");
        fp_t *d_a, *d_result;
        
        nvtxRangePush("Memory Allocation");
        CUDA_CHECK(hipMalloc(&d_a, n * sizeof(fp_t)));
        CUDA_CHECK(hipMalloc(&d_result, n * sizeof(fp_t)));
        nvtxRangePop();
        
        nvtxRangePush("Memory Copy H2D");
        CUDA_CHECK(hipMemcpy(d_a, h_a, n * sizeof(fp_t), hipMemcpyHostToDevice));
        nvtxRangePop();
        
        nvtxRangePush("Kernel Execution");
        int block_size = 256;
        int grid_size = (n + block_size - 1) / block_size;
        fp_inv_arrays_barrett<<<grid_size, block_size>>>(d_a, d_result, n);
        CUDA_CHECK(hipDeviceSynchronize());
        nvtxRangePop();
        
        nvtxRangePush("Memory Copy D2H");
        CUDA_CHECK(hipMemcpy(h_result, d_result, n * sizeof(fp_t), hipMemcpyDeviceToHost));
        nvtxRangePop();
        
        nvtxRangePush("Memory Deallocation");
        hipFree(d_a);
        hipFree(d_result);
        nvtxRangePop();
        nvtxRangePop();
    }
    
    void matrix_multiply(const fp_t* h_A, const fp_t* h_B, fp_t* h_C,
                        int m, int n, int k) {
        fp_t *d_A, *d_B, *d_C;
        
        CUDA_CHECK(hipMalloc(&d_A, m * n * sizeof(fp_t)));
        CUDA_CHECK(hipMalloc(&d_B, n * k * sizeof(fp_t)));
        CUDA_CHECK(hipMalloc(&d_C, m * k * sizeof(fp_t)));
        
        CUDA_CHECK(hipMemcpy(d_A, h_A, m * n * sizeof(fp_t), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_B, h_B, n * k * sizeof(fp_t), hipMemcpyHostToDevice));
        
        dim3 block_size(16, 16);
        dim3 grid_size((k + block_size.x - 1) / block_size.x,
                      (m + block_size.y - 1) / block_size.y);
        
        fp_matrix_mul_barrett<<<grid_size, block_size>>>(d_A, d_B, d_C, m, n, k);
        CUDA_CHECK(hipDeviceSynchronize());
        
        CUDA_CHECK(hipMemcpy(h_C, d_C, m * k * sizeof(fp_t), hipMemcpyDeviceToHost));
        
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }
};

// Example usage and testing
int main() {
    const fp_t p = 2147483647; // Large prime (2^31 - 1)
    FiniteFieldFpBarrett ff(p);
    
    const int n = 1000000;
    fp_t *a = new fp_t[n];
    fp_t *b = new fp_t[n];
    fp_t *result = new fp_t[n];
    
    // Initialize test data
    for (int i = 0; i < n; i++) {
        a[i] = i % p;
        b[i] = (i * 2) % p;
    }
    
    // Test addition
    ff.add_arrays(a, b, result, n);
    printf("Barrett Addition test: %u + %u = %u (mod %u)\n", a[0], b[0], result[0], p);
    
    // Test multiplication
    ff.mul_arrays(a, b, result, n);
    printf("Barrett Multiplication test: %u * %u = %u (mod %u)\n", a[0], b[0], result[0], p);
    
    // Test inverse (on smaller array for performance)
    const int small_n = 1000;
    ff.inv_arrays(a, result, small_n);
    printf("Barrett Inverse test: inv(%u) = %u (mod %u)\n", a[1], result[1], p);
    
    // Verify correctness by testing a[1] * result[1] ≡ 1 (mod p)
    fp_t verification[1] = {a[1]};
    fp_t inv_result[1] = {result[1]};
    fp_t verify_result[1];
    ff.mul_arrays(verification, inv_result, verify_result, 1);
    printf("Verification: %u * %u = %u (mod %u) [should be 1]\n", 
           a[1], result[1], verify_result[0], p);
    
    delete[] a;
    delete[] b;
    delete[] result;
    
    return 0;
}